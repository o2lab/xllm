#include "hip/hip_runtime.h"
#include "profiler.h"
#include<hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void
VectorAdd(
    const int *pA,
    const int *pB,
    int *pC,
    int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
    {
        pC[i] = pA[i] + pB[i];
    }
}


void
InitializeVector(
    int *pVector,
    int N)
{
    for (int i = 0; i < N; i++)
    {
        pVector[i] = i;
    }
}

static void
CleanUp(
    int *pHostA,
    int *pHostB,
    int *pHostC,
    int *pDeviceA,
    int *pDeviceB,
    int *pDeviceC)
{
    // Free host memory.
    if (pHostA)
    {
        free(pHostA);
    }
    if (pHostB)
    {
        free(pHostB);
    }
    if (pHostC)
    {
        free(pHostC);
    }

    // Free device memory.
    if (pDeviceA)
    {
        RUNTIME_API_CALL(hipFree(pDeviceA));
    }
    if (pDeviceB)
    {
        RUNTIME_API_CALL(hipFree(pDeviceB));
    }
    if (pDeviceC)
    {
        RUNTIME_API_CALL(hipFree(pDeviceC));
    }
}

void
DoVectorAddition()
{
    int N = 50000;
    size_t size = N * sizeof(int);
    int threadsPerBlock = 0;
    int blocksPerGrid = 0;
    int* pHostA, * pHostB, * pHostC;
    int* pDeviceA, * pDeviceB, * pDeviceC;
    int i, sum;

    // Allocate input vectors pHostA and pHostB in host memory.
    pHostA = (int *)malloc(size);
    MEMORY_ALLOCATION_CALL(pHostA);

    pHostB = (int *)malloc(size);
    MEMORY_ALLOCATION_CALL(pHostB);

    pHostC = (int *)malloc(size);
    MEMORY_ALLOCATION_CALL(pHostC);

    // Initialize input vectors
    InitializeVector(pHostA, N);
    InitializeVector(pHostB, N);
    memset(pHostC, 0, size);

    // Allocate vectors in device memory.
    RUNTIME_API_CALL(hipMalloc((void **)&pDeviceA, size));
    RUNTIME_API_CALL(hipMalloc((void **)&pDeviceB, size));
    RUNTIME_API_CALL(hipMalloc((void **)&pDeviceC, size));

    // Copy vectors from host memory to device memory.
    RUNTIME_API_CALL(hipMemcpy(pDeviceA, pHostA, size, hipMemcpyHostToDevice));
    RUNTIME_API_CALL(hipMemcpy(pDeviceB, pHostB, size, hipMemcpyHostToDevice));

    // Invoke kernel.
    threadsPerBlock = 256;
    blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    printf("Launching kernel: blocks %d, thread/block %d\n", blocksPerGrid, threadsPerBlock);

    VectorAdd <<< blocksPerGrid, threadsPerBlock >>> (pDeviceA, pDeviceB, pDeviceC, N);
    RUNTIME_API_CALL(hipGetLastError());

    // Copy result from device memory to host memory.
    // pHostC contains the result in host memory.
    RUNTIME_API_CALL(hipMemcpy(pHostC, pDeviceC, size, hipMemcpyDeviceToHost));

    // Verify result.
    for (i = 0; i < N; ++i)
    {
        sum = pHostA[i] + pHostB[i];
        if (pHostC[i] != sum)
        {
            fprintf(stderr, "Error: result verification failed\n");
            exit(EXIT_FAILURE);
        }
    }

    CleanUp(pHostA, pHostB, pHostC, pDeviceA, pDeviceB, pDeviceC);
}


int main(){
  CuptiProfiler profiler;
  profiler.start();
  DoVectorAddition();
  profiler.stop();
  return 0;
}
